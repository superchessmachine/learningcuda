//I am going to write something that will add vectors together


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] + b[i];
}

__managed__ int vector_a[257], vector_b[257], vector_c[257];

int main() {

	for(int i=0; i<257; i++) {
	
		vector_a[i]=i;

		vector_b[i]=256-i;

	}

	add<<<1,257>>>(vector_a, vector_b, vector_c);

	hipDeviceSynchronize();

	int result_sum=0;

	for (int i =0; i <257; i++) {
	
		result_sum += vector_c[i];
	}

	printf("Added vector\n");
	printf("Result: sum = %d\n", result_sum);

}
